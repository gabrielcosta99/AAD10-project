#include "hip/hip_runtime.h"
//
// Tomás Oliveira e Silva,  October 2024
//
// Arquiteturas de Alto Desempenho 2024/2025
//
// MD5 hash CUDA kernel code
//
// md5_cuda_kernel() --- each thread computes the MD5 hash of one message
//
// do not use this directy to search for DETI coins!
//

//
// needed stuff
//

typedef unsigned int u32_t;

#include "md5.h"

//
// the nvcc compiler stores x[] and state[] in registers (constant indices!)
//
// global thread number: n = threadIdx.x + blockDim.x * blockIdx.x
// global warp number: n >> 5
// warp thread number: n & 31
//

// typedef union{
//   u32_t coin_as_ints[13]; // 13 4-byte integers
//   char coin_as_chars[52]; // 52 = 13*4 bytes
// }coin_t;


extern "C" __global__ __launch_bounds__(128,1) void deti_coins_cuda_kernel_search(u32_t *deti_coins_storage_area, u32_t v1, u32_t v2)
{
    u32_t n,a,b,c,d,coin[13],hash[4],state[4],x[16], idx, n1,n2,n3;

    //
    // get the global thread number
    //
    n = (u32_t)threadIdx.x + (u32_t)blockDim.x * (u32_t)blockIdx.x;
    // printf("thread_number: %d\n",n);
    if(n <= 95*95*95){
        // if(n >9000)
        //     printf("n: %d\n",n);
        n3 = n/(95*95);
        n2 = n%(95*95) / 95 ;
        n1 = n % (95);
        // v1+= (n3 << 16) +(n2 << 8) + n1 ;
        // printf("n:%d, n2:%d,n1:%d,v2:%X, v1:%X\n",n,n2,n1,v2,v1);

        coin[0u] = 0x49544544; //  "DETI"
        coin[1u] = 0x696f6320; //  " coi"
        coin[2u] = 0x2020206E; //  "n   "
        coin[12u] = 0x0A202020; // "   \n"
        for(int i = 3u;i < 12u;i++)
            coin[i] = 0x20202020;
        coin[11u] += (n3 << 16) +(n2 << 8) + n1;
        coin[6] = v1;
        coin[7] = v2;
        for(int i = 0; i<95; i++){
//
// compute MD5 hash
//
# define C(c)         (c)
# define ROTATE(x,n)  (((x) << (n)) | ((x) >> (32 - (n))))
# define DATA(idx)    coin[idx]
# define HASH(idx)    hash[idx]
# define STATE(idx)   state[idx]
# define X(idx)       x[idx]
CUSTOM_MD5_CODE();
# undef C
# undef ROTATE
# undef DATA
# undef HASH
# undef STATE
# undef X


            // idx = atomicAdd(deti_coins_storage_area,13);
            // if(idx<=1000-13)
            //     ...deti_coins_storage_area[idx] = coin[0]
            if(hash[3u] == 0u)
            {
                idx = atomicAdd(deti_coins_storage_area,13);
                if(idx<=15000*13-13){
                    for(int j= 0;j<13;j++){
                        deti_coins_storage_area[idx+j] = coin[j];
                    }

                }
                // printf("idx: %3d,v2:%X,v1:%X\n",idx,v2,v1); // this value should be 1,14,27,etc...
            }
            coin[6u] += 1u;
            // coin[6u] += 1u << 16; // next combination
        }
    }
        
}

